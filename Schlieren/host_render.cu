#include "hip/hip_runtime.h"
/*
  For more information, please see: http://software.sci.utah.edu

  The MIT License

  Copyright (c) 2012-2013
  Scientific Computing and Imaging Institute, University of Utah

  License for the specific language governing rights and limitations under
  Permission is hereby granted, free of charge, to any person obtaining a
  copy of this software and associated documentation files (the "Software"),
  to deal in the Software without restriction, including without limitation
  the rights to use, copy, modify, merge, publish, distribute, sublicense,
  and/or sell copies of the Software, and to permit persons to whom the
  Software is furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included
  in all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
  OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
  THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
  FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
  DEALINGS IN THE SOFTWARE.
*/





#define DEBUG 0

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <teem/nrrd.h>


// includes, GL
#include "opengl_include.h"
#include <float.h>
#include <assert.h>

// includes
//#include "cutil.h"
//#include "cutil_math.h"
#include "hip/hip_runtime_api.h"
#include "helper_cuda_gl.h"
#include "helper_functions.h"
#include "hip/hip_vector_types.h"
#include <hip/hip_runtime.h>
#include "cuda_gl_interop.h"
#include <cstdlib>

#include "RenderParameters.h"
#include "kernel_render.h"
#include "kernel_filter.h"

RenderParameters* dparams;
hipArray* data_array = 0, *texture_array = 0, *color_array = 0;
unsigned int last_width, last_height;
unsigned int* d_out = 0;
float4* d_inout = 0;
float* d_rand_x = 0;

void Host_CopyMemory(RenderParameters* params);
void Host_Resize(RenderParameters* paramsp);

__host__ int rgbToIntHost(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

__host__ float getRandom()
{
    return drand48();
}



extern "C"
{

void Host_Init(RenderParameters* paramsp)
{
    hipMalloc((void**)&dparams, sizeof(RenderParameters));
    hipMemcpy(dparams, paramsp, sizeof(RenderParameters),hipMemcpyHostToDevice);
    //dparams = (RenderParameters*)malloc(sizeof(RenderParameters));

    //setup data texture
    tex_data.addressMode[0] = hipAddressModeClamp;
    tex_data.addressMode[1] = hipAddressModeClamp;
    tex_data.filterMode = hipFilterModeLinear;
    tex_data.normalized = false;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    hipExtent extent = make_hipExtent(paramsp->data_width, paramsp->data_height, paramsp->data_depth);
    checkCudaErrors( hipMalloc3DArray(&data_array, &channelDesc, extent) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)paramsp->data, extent.width*sizeof(float4), extent.width, extent.height);
    copyParams.dstArray = data_array;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    checkCudaErrors(  hipMemcpy3D(&copyParams) );

    hipBindTextureToArray(tex_data, data_array, channelDesc);

    /*
//setup data texture
tex_data2.addressMode[0] = hipAddressModeWrap;
tex_data2.addressMode[1] = hipAddressModeWrap;
tex_data2.filterMode = hipFilterModeLinear;
tex_data2.normalized = false;

hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  hipExtent const extent = make_hipExtent(paramsp->data_width, paramsp->data_height, paramsp->data_depth);
  checkCudaErrors( hipMalloc3DArray(&data_array, &channelDesc, extent) );
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr((void*)paramsp->data2, extent.width*sizeof(float), extent.width, extent.height);
  copyParams.dstArray = data_array;
  copyParams.extent = extent;
  copyParams.kind = hipMemcpyHostToDevice;
  checkCudaErrors(  hipMemcpy3D(&copyParams) );

  checkCudaErrors( hipBindTextureToArray(tex_data2, data_array, channelDesc) );
*/
    //setup cutoff texture
    if (paramsp->cutoff  == CUTOFF_IMAGE)
    {
        tex_cutoff.addressMode[0] = hipAddressModeClamp;
        tex_cutoff.addressMode[1] = hipAddressModeClamp;
        tex_cutoff.filterMode = hipFilterModeLinear;
        tex_cutoff.normalized = true;

        hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
        hipMallocArray(&texture_array, &channelDesc2, paramsp->cutoffSize.x, paramsp->cutoffSize.y);
        hipMemcpyToArray(texture_array, 0, 0, paramsp->cutoff_rgb, paramsp->cutoffSize.x*paramsp->cutoffSize.y*sizeof(float4), hipMemcpyHostToDevice);

        hipBindTextureToArray(tex_cutoff, texture_array, channelDesc2);
        paramsp->cutoff_dirty = false;
    }

    //
    //color testing code
    //
    /*
tex_color.addressMode[0] = hipAddressModeWrap;
tex_color.addressMode[1] = hipAddressModeWrap;
tex_color.filterMode = hipFilterModeLinear;
tex_color.normalized = false;

  hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
  extent = make_hipExtent(paramsp->data_width, paramsp->data_height, paramsp->data_depth);
  checkCudaErrors( hipMalloc3DArray(&color_array, &channelDesc3, extent) );
  hipMemcpy3DParms copyParams3 = {0};
  copyParams3.srcPtr = make_hipPitchedPtr((void*)paramsp->color_data, extent.width*sizeof(float4), extent.width, extent.height);
  copyParams3.dstArray = color_array;
  copyParams3.extent = extent;
  copyParams3.kind = hipMemcpyHostToDevice;
  checkCudaErrors(  hipMemcpy3D(&copyParams3) );

  hipBindTextureToArray(tex_color, color_array, channelDesc);
*/

    Host_Resize(paramsp);



}


void Host_Render(RenderParameters* paramsp)
{
    // printf("rendering...");
    if (last_width != paramsp->width || last_height != paramsp->height)
        Host_Resize(paramsp);
    //for(size_t i = 0; i < paramsp->width*paramsp->height; i++)
    //        paramsp->inout_rgb[i] = make_float4(0,0,0,0);
    RenderParameters& params = *paramsp;
    Host_CopyMemory(paramsp);
    dim3 block(16,16,1);
    dim3 grid((params.width/block.x),(params.height/block.y),1);   //positions go 0 to 100, which maps to -1 to 1 on each lightplace axis
    //float3 lookup_scale = {1.0f/(params.max_bound.x-params.min_bound.x), 1.0f/(params.max_bound.y - params.min_bound.y), 1.0f/(params.max_bound.z-params.min_bound.z)};
    paramsp->rand1 = drand48();
    paramsp->rand2 = drand48();
    paramsp->rand3 = drand48();
    paramsp->rand4 = drand48();
    // allocate space on the device for the results
    hipMemcpy(dparams, paramsp, sizeof(RenderParameters),hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    for(int i =0; i < paramsp->numRenderPasses; i++)
    {
        kernel_render<<< grid, block>>>(dparams, d_inout, d_out, d_rand_x);
        hipDeviceSynchronize();
        paramsp->passes+=paramsp->raysPerPixel;
    }
    // hipMemcpy(paramsp->out_rgb, d_out, sizeof(unsigned int)*params.width*params.height, hipMemcpyDeviceToHost);
    //    hipMemcpy(paramsp->inout_rgb, d_inout, sizeof(float4)*params.width*params.height, hipMemcpyDeviceToHost);
    //for(int i = 0 ; i < params.width*params.height; i++)
    // {
    // paramsp->out_rgb[i] = 0;
    //}
    hipDeviceSynchronize();
    kernel_filter<<< grid, block>>>(dparams, d_inout, d_out);
    hipMemcpy(paramsp->out_rgb, d_out, sizeof(unsigned int)*params.width*params.height, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    /*        float m = 255.0/paramsp->passes;
for(int i = 0; i < paramsp->width; i++)
{
    for(int j = 0; j < paramsp->height; j++)
    {
        size_t index = i + j*paramsp->width;
        float4 c = paramsp->inout_rgb[index];
        //c =  make_float4(1.0,1.0,1.0,1.0);
        paramsp->out_rgb[index] = rgbToIntHost(c.x*m,c.y*m,c.z*m);
    }
}*/

    for(int i = 0; i < paramsp->width; i++)
    {
        for(int j = 0; j < paramsp->height; j++)
        {
            size_t index = i + j*paramsp->width;
            unsigned int c = paramsp->out_rgb[index];
            unsigned char* ca = (unsigned char*)&c;
            ca[3] = 255;
            paramsp->out_rgb[index] = c;
        }
    }
    glDrawPixels(params.width, params.height, GL_RGBA, GL_UNSIGNED_BYTE, paramsp->out_rgb);
    // printf(" rendering finished.\n");
}

void Host_Clear(RenderParameters* paramsp)
{
    if (!d_inout)
        return;
    hipMemcpy(d_inout, paramsp->inout_rgb, sizeof(float4)*paramsp->width*paramsp->height, hipMemcpyHostToDevice);
}

void Host_Kill()
{

    RenderParameters* dparams;
    hipArray* data_array = 0, *texture_array = 0, *color_array = 0;
    unsigned int last_width, last_height;
    unsigned int* d_out = 0;
    float4* d_inout = 0;
    float* d_rand_x = 0;
    hipFree(d_inout);
    hipFree(dparams);
    hipFree(d_out);
    hipFree(d_rand_x);

    hipUnbindTexture (tex_data);
    checkCudaErrors (hipFreeArray (data_array));
    hipUnbindTexture (tex_data2);
    checkCudaErrors (hipFreeArray (texture_array));
    hipUnbindTexture (tex_cutoff);
    checkCudaErrors (hipFreeArray (color_array));
}


}

void Host_CopyMemory(RenderParameters* paramsp)
{
    //TODO: NOTE: for debugging perposes only memcopy, however need to support size changes

    if (paramsp->cutoff_dirty)
    {
        //if (texture_array)
        //    hipFree(texture_array);
        hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
        //hipMallocArray(&texture_array, &channelDesc2, paramsp->cutoffSize.x, paramsp->cutoffSize.y);
        hipMemcpyToArray(texture_array, 0, 0, paramsp->cutoff_rgb, paramsp->cutoffSize.x*paramsp->cutoffSize.y*sizeof(float4), hipMemcpyHostToDevice);
        paramsp->cutoff_dirty = false;
    }
}

void Host_Resize(RenderParameters* paramsp)
{
    printf("resizing to %d %d \n", paramsp->width, paramsp->height);
    paramsp->passes = 0;
    int window_size = paramsp->width*paramsp->height;
    if (d_inout)
        hipFree(d_inout);
    hipMalloc((void**)&d_inout, sizeof(float4)*paramsp->width*paramsp->height);
    if (paramsp->inout_rgb)
        delete [] paramsp->inout_rgb;
    paramsp->inout_rgb = new float4[paramsp->width*paramsp->height];
    for(size_t i = 0; i < paramsp->width*paramsp->height; i++)
        paramsp->inout_rgb[i] = make_float4(0,0,0,0);
    hipMemcpy(d_inout, paramsp->inout_rgb, sizeof(float4)*paramsp->width*paramsp->height, hipMemcpyHostToDevice);

    if (d_out)
        hipFree(d_out);
    hipMalloc((void**)&d_out, sizeof(unsigned int)*paramsp->width*paramsp->height);
    last_width = paramsp->width;
    last_height = paramsp->height;
    if (paramsp->out_rgb)
        free(paramsp->out_rgb);
    paramsp->out_rgb = (unsigned int*)malloc(sizeof(unsigned int)*paramsp->width*paramsp->height);

    if (d_rand_x)
        hipFree(d_rand_x);

    paramsp->random_array  = (float*)malloc(sizeof(float)*window_size);
    for(int i =0;i<window_size;i++){
        paramsp->random_array[i] = getRandom();
    }
    hipMalloc((void**)&d_rand_x, window_size*sizeof(float));
    hipMemcpy( d_rand_x, paramsp->random_array,sizeof(float)*window_size, hipMemcpyHostToDevice  );
}

